
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <omp.h>
#include <stdio.h>
using namespace std;
int dim;
string arquivo = "grafo2500.txt";
int retornaDimensao(){
  ifstream arq (arquivo.c_str(),std::ifstream::in);
  int d = 0;
  if(arq.is_open()){
    arq >> d;
  }
  arq.close();
  return d;
}

//função para descobrir se existe 0 no vetor de cores (caso exista 0, é porque tem ainda de colorir o vértice)
bool ha_elementos_a_colorir (int vet[], int tam){
  for (int i=0; i< tam; i++){
    if (vet[i] == 0 || vet[i] == -1){
      return true;
    }
  }
  return false;
}

bool possuiAresta (int vet[], int tam){
  for (int i=0; i< tam; i++){
    if (vet[i] == 1){
      return true;
    }
  }
  return false;
}

int menor_grau(int a[],int tam, int* vetcores){
  int grau;
  int menor = tam;
  bool primeiro = true;
  int vertice_menor_grau;
  for(int i = 0; i < tam; i++){
    grau = 0;
    for(int j = 0; j < tam; j++){
      grau = grau + a[j + i*dim];
    }
    if((primeiro || grau < menor) && vetcores[i] == 0){
      primeiro = false;
      menor = grau;
      vertice_menor_grau =i;
    }
  }
  return vertice_menor_grau;
}

int* retornaMatriz(){

  int *a;
  ifstream arq(arquivo.c_str(),std::ifstream::in);
  if(arq.is_open()){
    arq >> dim;
        //cout << "Dimensao: "<< dimensao <<endl;
    a = new int[dim*dim];
    for(int i=0;i<dim*dim;i++){
      arq >> a[i];
    }
  }
  arq.close();
  return a;
}


__device__
void zera_aresta(int *a,int dim, int v, int* d_vetcores){

  if(d_vetcores[v] == 0){
    d_vetcores[v] = -1;
  }
  for(int i = 0; i < dim; i++){
    if(i == v){
      for(int j = 0; j < dim; j++){
        a[j + i*dim] = 0;
      }
    }
  }
}

__global__
void d_main(int dim, int* d_vetcores, int* w, int* u, int vzin, int num_threads){
  int aux = blockDim.x * blockIdx.x + threadIdx.x;
  int i;
  for(i = aux; i < dim; i += blockDim.x * num_threads){
    if(vzin == i){
      for(int j =0; j < dim; j++){
        if(w[j +i*dim] == 1){
          zera_aresta(w,dim,j,d_vetcores);
        }
      }
      zera_aresta(w,dim,i,d_vetcores);
      zera_aresta(u,dim,i,d_vetcores);
    }
    __syncthreads();
  }
}

int main(){
    //variaveis da cpu
  int *u = retornaMatriz();
  int *vetcores = new int[dim];
  //variaveis da gpu
  int *d_u, *d_vetcores, *d_w;
  int *w = new int[dim*dim];
  int vzin;
  int cor_atual = 1;
  //alocando acesso pra gpu
  hipMalloc(&d_vetcores,dim*sizeof(int));
  hipMalloc(&d_u,       dim*dim*sizeof(int));
  hipMalloc(&d_w,       dim*dim*sizeof(int));

  int num_threads = 16;
  int num_blocks  = 32;
  clock_t tStart = clock();
  double tfor = 0;
  for (int i=0; i<dim; i++){
    vetcores[i] = 0;
  }
  while (ha_elementos_a_colorir(vetcores, dim)){
        memcpy(w, u, dim*dim*sizeof(int));
        hipMemcpy(d_w, w, dim*dim*sizeof(int), hipMemcpyHostToDevice);// w = u
        hipMemcpy(d_u, w, dim*dim*sizeof(int), hipMemcpyHostToDevice);// w = u
        for(int i = 0; i < dim; i++){
          if(vetcores[i] == -1){
            vetcores[i] = 0;
          }
        }
        while(possuiAresta(w,dim*dim)){
            vzin = menor_grau(w,dim, vetcores);// acha o vertice de menor grau em w
            vetcores[vzin] = cor_atual ;// colore
            hipMemcpy(d_vetcores, vetcores, dim*sizeof(int),hipMemcpyHostToDevice);
            tStart = clock();
            d_main<<<num_threads,num_blocks>>>(dim, d_vetcores, d_w, d_u, vzin, num_threads);
            tfor +=  ((double)(clock() - tStart)/CLOCKS_PER_SEC);
            hipMemcpy(vetcores, d_vetcores, dim*sizeof(int),hipMemcpyDeviceToHost);
            hipMemcpy(w, d_w, dim*dim*sizeof(int), hipMemcpyDeviceToHost);// w = u
            hipMemcpy(u, d_u, dim*dim*sizeof(int), hipMemcpyDeviceToHost);// w = u

          //   cout << "vetcores: ";
          //   for (int i = 0; i < dim; ++i)
          //   {
          //       cout << vetcores[i] << ", ";
          //   }
          //   cout  << endl;

          //   cout << "w: \n";
          //   for (int i = 0; i < dim*dim; ++i)
          //   {
          //       cout << w[i] << ", ";
          //       if(i%dim == dim-1)
          //           cout << endl;
          //   }
          //   cout  << endl;

          //   cout << "u: \n";
          //   for (int i = 0; i < dim*dim; ++i)
          //   {
          //       cout << u[i] << ", ";
          //       if(i%dim == dim-1)
          //           cout << endl;
          //   }
          //   cout  << endl;
          }
          cor_atual++;
        }
        for(int i = 0; i < dim; i++){
          // cout << "vetcores[" << i << "]: " << vetcores[i] << endl;
        }
        cout << tfor;
        return 0;

      }