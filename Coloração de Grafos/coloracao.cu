
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <omp.h>

using namespace std;

int dim;
int* retornaMatriz(){

    int *a;
    ifstream arq("grafo.txt",std::ifstream::in);
    if(arq.is_open()){
        arq >> dim;
        //cout << "Dimensao: "<< dimensao <<endl;
        a = new int[dim*dim];
        for(int i=0;i<dim*dim;i++){
            arq >> a[i];
        }
    }
    arq.close();
    return a;
}

__device__
void zera_aresta(int *a, int dim, int *vetcores, int v){// ok
  if(vetcores[v] == 0){
    vetcores[v] = -1;
}
    for(int i = 0; i < dim; i++){
        if(i == v){
            for(int j = 0; j < dim; j++){
                a[j + i*dim] = 0;
            }
        }
    }
}

__device__
bool possui_aresta(int vet[], int tam){
  for (int i=0; i< tam; i++){
        if (vet[i] == 1){
            return true;
        }
    }
    return false;
}

__device__
bool ha_elementos_a_colorir(int vet[], int tam){// ok
  for (int i=0; i< tam; i++){
        if (vet[i] == 0 || vet[i] == -1){
            return true;
        }
    }
    return false;
}

__global__
void teste_kernel(int dim, int *u, int *vetcores){

int cor_atual = 1;// iniciando c a primeira cor valendo 1
int vzin;// vertice de menor grau
int idx = __umul24( blockDim.x, blockIdx.x) + threadIdx.x;
zera_aresta(u,dim,vetcores,0);
  for (int i = 0; i < dim; i++){
     for(int j = 0; j < dim; j++){
       if(u[j + i*dim] == 1)
        u[j + i*dim] = idx;
     }
  }

//iniciando o vetor de cores
  for (int i = 0; i < dim; i ++){

    vetcores[i] = 0;
  }


}


int main(){

  //variaveis da cpu
  int *u = retornaMatriz();
  int *vetcores = new int[dim];
  //variaveis da gpu
  int *d_u, *d_vetcores, *d_w;

  //iniciando o vetor de cores
  for (int i=0; i<dim; i++){
        vetcores[i] = 0;
    }
  //alocando acesso pra gpu
  hipMalloc(&d_vetcores,dim*sizeof(int));
  hipMalloc(&d_u,       dim*dim*sizeof(int));
  hipMalloc(&d_w,       dim*dim*sizeof(int));
  //copiando a matriz pra gpu
  hipMemcpy(d_u,u,dim*dim*sizeof(int),hipMemcpyHostToDevice);
  //executando função da gpu
  teste_kernel<<<16,16>>>(dim,d_u,d_vetcores);
  //esperando acabar o processamento da gpu
  hipDeviceSynchronize();
  //pegando o resultado da gpu e jogando pro processador
  hipMemcpy(u,d_u,dim*dim*sizeof(int),hipMemcpyDeviceToHost);
  hipMemcpy(vetcores,d_vetcores,dim*sizeof(int),hipMemcpyDeviceToHost);


  for (int i = 0; i < dim; i++){
   for(int j = 0; j < dim; j++){
     cout << u[j + i*dim] << " ";
   }
   cout << endl;
}

for (int i=0; i<dim; i++){
        cout << i << " " << vetcores[i] << endl;
    }

  hipFree(d_u);
  hipFree(d_vetcores);

  return 0;

}